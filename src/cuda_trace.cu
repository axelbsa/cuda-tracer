#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>

#include <hiprand/hiprand_kernel.h>

#include "cuda_helpers/hip/hip_runtime_api.h"
#include "cuda_helpers/hip/hip_vector_types.h"

#include "raytracer.h"

__device__ float3 color(const Ray& r, Hittable **world, hiprandState *local_rand_state) {
    Ray cur_ray = r;
    float3 cur_attenuation = make_float3(1.0f, 1.0f, 1.0f);
    for (int i = 0; i < 50; ++i) {
        hit_record rec;
        float3 attenuation;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            if (rec.mat_prt->scatter(r, rec, attenuation, cur_ray, local_rand_state))
            {
                cur_attenuation *= attenuation;
            }
        } else {
            float3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y + 1.0f);
            float3 c = (1.0f - t) * make_float3(1.0f, 1.0f, 1.0f) + t * make_float3(0.5f, 0.7f, 1.0f);
            return cur_attenuation * c;
        }
    }
    return make_float3(0.0f, 0.0f, 0.0f);
}

__global__ void d_render(
        uchar4 *output, uint imageW, uint imageH,
        Hittable **world, Camera **d_camera, hiprandState *d_rand_state)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    float3 col = make_float3(0.0f, 0.0f, 0.0f);

    if((x >= imageW) || (y >= imageH))
        return;

    uint i = y * imageW + x;  // Array transformation to 1D?
    hiprandState local_rand_state = d_rand_state[i];
#define ns 1
    for(int s=0; s < ns; s++) {
        float u = float(x + hiprand_uniform(&local_rand_state)) / float(imageW);
        float v = float(y + hiprand_uniform(&local_rand_state)) / float(imageH);
        Ray r = (*d_camera)->get_ray(u,v);
        col += color(r, world, &local_rand_state);
    }
    d_rand_state[i] = local_rand_state;
    col /= float(ns);
    //printf("Color contrib: %f %f %f\n",col.x, col.y, col.z);
    output[i] = to_uchar4( make_float4( col, 1.0 ) * 255.99 );
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {

   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;

   if((i >= max_x) || (j >= max_y))
       return;

   int pixel_index = j * max_x + i;

   //Each thread gets same seed, a different sequence number, no offset
   hiprand_init(j * 1984 + i, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void create_world(dim3 windowSize, Hittable **d_list, Hittable **d_world, Camera **d_camera)
{
    // Make sure we only do this once
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        Material *mat_small_lamb = new Lambertian(make_float3(0.8f, 0.3f, 0.3f));
        Material *mat_big_lamb = new Lambertian(make_float3(0.8f, 0.8f, 0.0f));
        Material *mat_small_met_1 = new Metal(make_float3(0.8f, 0.6f, 0.2f), 0.3f);
        Material *mat_small_met_2 = new Metal(make_float3(0.8f, 0.8f, 0.8f), 1.0f);

        *(d_list + 0) = new Sphere(make_float3(0.0f, 0.0f, -1.0f), 0.5f, mat_small_lamb);
        *(d_list + 1) = new Sphere(make_float3(0.0f, -100.5f,-1.0f), 100.0f, mat_big_lamb);
        *(d_list + 2) = new Sphere(make_float3(1.0f, 0.0f,-1.0f), 0.5f, mat_small_met_1);
        *(d_list + 3) = new Sphere(make_float3(-1.0f, 0.0f,-1.0f), 0.5f, mat_small_met_2);

        *d_world = new Hittable_list(d_list, 4);
        *d_camera = new Camera(windowSize);
    }
}

extern "C" void init_cuda_scene(
        dim3 windowSize,
        Hittable **d_list,
        Hittable **d_world,
        Camera **d_camera)
{

    create_world <<<1, 1>>> (windowSize, d_list, d_world, d_camera);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Remember to free somewhere
    //checkCudaErrors(hipFree(d_camera));
    //checkCudaErrors(hipFree(d_world));
    //checkCudaErrors(hipFree(d_list));
}

extern "C" void init_cuda_rng_state(
        dim3 windowSize,
        dim3 threads,
        hiprandState *d_rand_state)
{

    render_init<<<windowSize, threads>>>(windowSize.x, windowSize.y, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

extern "C" void trace(
        dim3 blocks,
        dim3 threads,
        uchar4 *d_output,
        uint imageW,
        uint imageH,
        hiprandState *d_rand_state,
        Hittable **d_world,
        Camera **d_camera)
{
    d_render<<<blocks, threads>>>(
            d_output, imageW, imageH, d_world, d_camera, d_rand_state);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}
